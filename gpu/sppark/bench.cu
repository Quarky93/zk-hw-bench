#include "hip/hip_runtime.h"
#include <vector>
#include <thread>
#include <chrono>
#include <iostream>
#include <bls12-381.hpp>

#ifndef N_THREADS
#define N_THREADS 16384
#endif
#define N_ITERATIONS 10000000L
#define WARP_SIZE 32

__global__ void mod_sqr(fp_t res[N_THREADS]) {
    const unsigned long idx = blockIdx.x * blockDim.x + threadIdx.x;
    const uint64_t limbs[6] = {123456789 + idx, 0, 0, 0, 0, 0};
    fp_t x = *((fp_t *) limbs);
    for (long i = 0; i < N_ITERATIONS; i++) {
        x = x * x;
    }
    res[idx] = x;
}

int main() {
    fp_t *h_res = (fp_t *) malloc(sizeof(fp_t) * N_THREADS);
    fp_t *d_res;
    hipMalloc(&d_res, sizeof(fp_t) * N_THREADS);

    auto t_start = std::chrono::high_resolution_clock::now();
    mod_sqr<<<N_THREADS/WARP_SIZE, WARP_SIZE>>>(d_res);
    hipDeviceSynchronize();
    auto t_end = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(t_end - t_start);
    hipMemcpy(h_res, d_res, sizeof(fp_t) * N_THREADS, hipMemcpyDeviceToHost);
    std::cout << "Elapsed time: " << elapsed.count() << " ms\n";
    std::cout << "Throughput: " << (N_THREADS * N_ITERATIONS / 1000000000.0) / (elapsed.count() / 1000.0) << " GOPS\n";

    return 0;
}
